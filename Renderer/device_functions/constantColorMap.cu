#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix.h>

#include "LaunchParams.h"
#include "perRayData.h"
#include "shaderUtils.h"

namespace nert_renderer {

    extern "C" __device__ float3 __direct_callable__sampleEnvironmapLight(const light_data & envmap, Random & random) {
        float phi = random() * 2 * M_PIf;
        float cosTheta = random() * 2 - 1;
        float sinTheta = sqrtf(1.0f - cosTheta * cosTheta);
        return make_float3(sinTheta * cos(phi), sinTheta * sin(phi), cosTheta);
    }

    // Don't multiply by solid angle;
    // the PDF was already multiplied by solid angle during construction
    extern "C" __device__ float __direct_callable__pdf(const light_data & envmap, const float3 & direc) {
        return 1 / (4 * M_PIf);
    }

    extern "C" __device__ float3 __direct_callable__evaluate(const light_data & envmap, const float3 & direc) {
        return envmap.color;
    }
}