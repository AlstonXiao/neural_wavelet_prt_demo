#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix.h>

#include "LaunchParams.h"
#include "perRayData.h"
#include "shaderUtils.h"
#define RR_DEPTH 3

namespace nert_renderer
{
    extern "C" __constant__ LaunchParams optixLaunchParams;

    static __forceinline__ __device__ void traceRadiance(
        OptixTraversableHandle handle,
        float3 ray_origin,
        float3 ray_direction,
        PRD &prd)
    {
        unsigned int
            u0,
            u1, u2,        // Origin (R)
            u3, u4, u5,    // Direction (R)
            u6, u7, u8,    // pixelcolor (R)
            u9,            // Ray Recursive depth (W)
            u10, u11, u12, // Light contribution (RW)
            u13,           // Random (RW)
            u14,           // PDF (RW)
            u15,           // Flags (R)
            u16, u17;      // Pointer to extra (W)

        u6 = __float_as_uint(prd.pixelColor.x);
        u7 = __float_as_uint(prd.pixelColor.y);
        u8 = __float_as_uint(prd.pixelColor.z);
        u9 = prd.ray_recursie_depth;
        u10 = __float_as_uint(prd.light_contribution.x);
        u11 = __float_as_uint(prd.light_contribution.y);
        u12 = __float_as_uint(prd.light_contribution.z);
        u13 = prd.random.state;
        u14 = __float_as_uint(prd.pdf);

        u15 = prd.firstHitUpdated | prd.pixelDirectSampleHitEnvMapFlag << 1 |
              prd.pixelPerfectReflectionFlag << 2 | prd.done << 3;

        const uint64_t uptr = reinterpret_cast<uint64_t>(prd.prdExtraInfo);
        u16 = uptr >> 32;
        u17 = uptr & 0x00000000ffffffff;

        optixTrace(
            PAYLOAD_TYPE_RADIANCE,
            handle,
            ray_origin,
            ray_direction,
            1e-2f, // tmin
            1e20f, // tmax
            0.0f,  // rayTime
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
            RADIANCE_RAY_TYPE,             // SBT offset
            RAY_TYPE_COUNT,                // SBT stride
            RADIANCE_RAY_TYPE,             // missSBTIndex
            u0, u1, u2, u3, u4, u5, u6, u7, u8, u9, u10, u11, u12, u13, u14, u15, u16, u17);

        prd.ray_origin = make_float3(__uint_as_float(u0), __uint_as_float(u1), __uint_as_float(u2));
        prd.ray_dir = make_float3(__uint_as_float(u3), __uint_as_float(u4), __uint_as_float(u5));
        prd.pixelColor = make_float3(__uint_as_float(u6), __uint_as_float(u7), __uint_as_float(u8));

        prd.light_contribution = make_float3(__uint_as_float(u10), __uint_as_float(u11), __uint_as_float(u12));
        prd.random.state = u13;
        prd.pdf = __uint_as_float(u14);

        prd.firstHitUpdated = u15 & 1u;
        prd.pixelDirectSampleHitEnvMapFlag = (u15 & (1u << 1)) >> 1;
        prd.pixelPerfectReflectionFlag = (u15 & (1u << 2)) >> 2;
        prd.done = (u15 & (1u << 3)) >> 3;
    }

    extern "C" __global__ void __raygen__SHRenderCamera()
    {

        // compute a test pattern based on pixel ID
        const int ix = optixGetLaunchIndex().x;
        const int iy = optixGetLaunchIndex().y;

        const auto &camera = optixLaunchParams.camera;

        PRDExtra prdE;
        prdE.cleanForNextLaunch();
        prdE.ix = ix;
        prdE.iy = iy;

        // frame ID will make the realtime visualization constant
        PRD prd(ix + optixLaunchParams.frame.size.x * iy, optixLaunchParams.frame.frameID, &prdE);

        float2 screen;
        screen = make_float2(ix + prd.random() - 0.5 - optixLaunchParams.camera.cx, iy + prd.random() - 0.5 - optixLaunchParams.camera.cy);
        screen = screen / make_float2(optixLaunchParams.camera.fx, optixLaunchParams.camera.fy);

        // generate ray direction
        float3 initialRayOrigin = (float3)camera.position;
        float3 initialRayDir = (float3)normalize(camera.direction
            + screen.x * camera.horizontal
            + screen.y * camera.vertical);

        prdE.cleanForNextLaunch();
        prd.cleanForNextLaunch();

        float3 currentRayOrigin = initialRayOrigin;
        float3 currentRayDir = initialRayDir;
        traceRadiance(optixLaunchParams.traversable, currentRayOrigin, currentRayDir, prd);

        uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;

        float3 pixelNormal = normalize(prdE.pixelNormalFirstHit);
        float3 worldNormal(pixelNormal / 2 + make_float3(0.5));
        // move to camera space
        float3 cameraNormal = normalize(to_local(normalize(pixelNormal), normalize(camera.direction), normalize(camera.horizontal)));
        cameraNormal = cameraNormal / 2 + make_float3(0.5);
        if (!isnan(prd.pixelColor.x) && !isnan(prd.pixelColor.y) && !isnan(prd.pixelColor.z)) {
            float3 accum = prd.pixelColor;
            if (optixLaunchParams.frame.frameID > 0) {
                accum += float(optixLaunchParams.frame.frameID)
                    * make_float3(optixLaunchParams.frame.colorBuffer[fbIndex]);
                accum /= float(optixLaunchParams.frame.frameID + 1.f);
            }
            optixLaunchParams.frame.colorBuffer[fbIndex] = make_float4(accum, 1.f);

        }

        optixLaunchParams.frame.cameraNormalBuffer[fbIndex] = make_float4(cameraNormal, 1.f);
        optixLaunchParams.frame.worldNormalBuffer[fbIndex] = make_float4(worldNormal, 1.f);

        optixLaunchParams.frame.FirstHitKdBuffer[fbIndex] = make_float4(prdE.pixelFirstHitKd, 1.f);
        optixLaunchParams.frame.FirstHitKsBuffer[fbIndex] = make_float4(prdE.pixelFirstHitKs, 1.f);
    }
}